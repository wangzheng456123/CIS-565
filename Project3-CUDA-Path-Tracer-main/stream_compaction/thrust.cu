#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
            int* dev_idata;
            int* dev_odata;

            hipMalloc((void**) & dev_idata, n * sizeof(int));
            hipMalloc((void**) & dev_odata, n * sizeof(int));

            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);

            thrust::device_ptr<int> thrust_idata(dev_idata);
            thrust::device_ptr<int> thrust_odata(dev_odata);

            thrust::exclusive_scan(thrust_idata, thrust_idata + n, thrust_odata);

            hipMemcpy(odata, dev_odata, sizeof(int) * n, hipMemcpyDeviceToHost);

            hipFree(dev_idata);
            hipFree(dev_odata);

            timer().endGpuTimer();
        }
    }
}
